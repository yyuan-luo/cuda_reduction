
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void reduce1(int *g_idata, int *g_odata) {
   extern __shared__ int sdata[];

   unsigned int tid = threadIdx.x;
   unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
   sdata[tid] = g_idata[tid];
   __syncthreads();

   for (unsigned int s = 1; s < blockDim.x; s *= 2) {
      if (tid % (2*s) == 0) {
       sdata[tid] += sdata[tid + s];
      }
      __syncthreads();
   }
   
   if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void reduce2(int *g_idata, int *g_odata) {
   extern __shared__ int sdata[];

   unsigned int tid = threadIdx.x;
   unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
   sdata[tid] = g_idata[tid];
   __syncthreads();

   for (unsigned int s = 1; s < blockDim.x; s *= 2) {
      int index = 2 * s * tid;
      if (index < blockDim.x) {
       sdata[index] += sdata[index + s];
      }
      __syncthreads();
   }
   
   if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

int main() {

}